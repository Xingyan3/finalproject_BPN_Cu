#include <stdio.h>
#include <stdlib.h>
#include "support.h"
#include "propagate.cu"
#include "backpropagate.cu"

int main(int argc, char *argv[])
{
  NET  Net;
  int Stop;
  double MinTestError;

  InitializeRandoms();
  GenerateNetwork(&Net);
  RandomWeights(&Net);
  InitializeApplication(&Net);

  hipMalloc((void**)&Net, sizeof(NET)*1);
  hipDeviceSynchronize();

  Stop = FALSE;
  MinTestError = MAX_REAL;
  do {
    TrainNet(&Net, 10);
    TestNet(&Net);
    if (TestError < MinTestError) {
      fprintf(f, " - saving Weights ...");
      MinTestError = TestError;
      SaveWeights(&Net);
    }
    else if (TestError > 1.2 * MinTestError) {
      fprintf(f, " - stopping Training and restoring Weights ...");
      Stop = TRUE;
      RestoreWeights(&Net);
    }
  } while (NOT Stop);

  TestNet(&Net);
  EvaluateNet(&Net);
   
  FinalizeApplication(&Net);
}