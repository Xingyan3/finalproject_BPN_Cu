#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include.h"

#define TILE_SIZE 16

__global__ void PropagateLayerKernel(REAL* layerOutput, REAL* nextLayerOutput, REAL* weight, const REAL gain, const int units, const int nextUnits)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < nextUnits) {
        REAL sum = 0;
        for (int j = 0; j < units; j++) {
            sum += weight[i * units + j] * layerOutput[j];
        }
        nextLayerOutput[i] = 1 / (1 + exp(-gain * sum));
    }
}


void PropagateNetCUDA(NET *Net, NET *Net_d, int NUM_LAYERS)
{
    int blockSize = TILE_SIZE;

    for (int l = 0; l < NUM_LAYERS - 1; l++)
    {
        int units = Net->Layer[l]->Units;
        int nextUnits = Net->Layer[l + 1]->Units;
        int numBlocks = (nextUnits + blockSize - 1) / blockSize;

        int size = nextUnits * units;
    
        REAL *d_weight, *d_layerOutput, *d_nextLayerOutput;
        hipMalloc((REAL**)&d_weight, size * sizeof(REAL));
        hipMalloc((REAL**)&d_layerOutput, units * sizeof(REAL));
        hipMalloc((REAL**)&d_nextLayerOutput, nextUnits * sizeof(REAL));

        hipMemcpy(d_layerOutput, Net->Layer[l]->Output, units * sizeof(REAL), hipMemcpyHostToDevice);
        hipMemcpy(d_weight, Net->Layer[l + 1]->Weight, size * sizeof(REAL), hipMemcpyHostToDevice);

        PropagateLayerKernel<<<numBlocks, blockSize>>>(d_layerOutput, d_nextLayerOutput, d_weight, Net->Gain, units, nextUnits);

        hipMemcpy(Net->Layer[l + 1]->Output, d_nextLayerOutput, nextUnits * sizeof(REAL), hipMemcpyDeviceToHost);
            
        hipFree(d_layerOutput);
        hipFree(d_nextLayerOutput);
        hipFree(d_weight);
    }
}


__global__ void BackpropagateLayerKernel(REAL* output, REAL* error, REAL* weight, const REAL gain, int units, int prevUnits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < units) {
        REAL out = output[i];
        REAL err = 0;
        for (int j = 0; j < prevUnits; j++) {
            err += weight[j * units + i] * error[j];
        }
        error[i] = gain * out * (1 - out) * err;
    }
}

__global__ void AdjustWeightsKernel(REAL* output, REAL* error, REAL* weight, REAL* dWeight, const REAL eta, const REAL alpha, const int units, const int prevUnits) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < units) {
        for (int j = 0; j < prevUnits; j++) {
            REAL out = output[j];
            REAL err = error[i];
            REAL dW = dWeight[i * prevUnits + j];
            weight[i * units + j] += eta * err * out + alpha * dW;
            dWeight[i * units + j] = eta * err * out;
        }
    }
}

void BackpropagateNetCUDA(NET *Net, int NUM_LAYERS)
{
    int blockSize = TILE_SIZE;

    for (int l = NUM_LAYERS - 1; l > 1; l--)
    {
        int units = Net->Layer[l]->Units;
        int prevUnits = Net->Layer[l - 1]->Units;

        int numBlocks = (prevUnits + blockSize - 1) / blockSize;

        int size = prevUnits * (units+1);
    
        REAL *d_weight, *d_dweight, *d_prevlayerOutput, *d_prevLayerError, *d_LayerError;
        hipMalloc((REAL**)&d_weight, size * sizeof(REAL));
        hipMalloc((REAL**)&d_dweight, size * sizeof(REAL));
        hipMalloc((REAL**)&d_prevlayerOutput, prevUnits * sizeof(REAL));
        hipMalloc((REAL**)&d_prevLayerError, prevUnits * sizeof(REAL));
        hipMalloc((REAL**)&d_LayerError, units * sizeof(REAL));

        hipMemcpy(d_prevLayerError, Net->Layer[l - 1]->Output, prevUnits * sizeof(REAL), hipMemcpyHostToDevice);
        hipMemcpy(d_prevlayerOutput, Net->Layer[l - 1]->Output, prevUnits * sizeof(REAL), hipMemcpyHostToDevice);
        hipMemcpy(d_weight, Net->Layer[l]->Weight, size * sizeof(REAL), hipMemcpyHostToDevice);

        BackpropagateLayerKernel<<<numBlocks, blockSize>>>(d_prevlayerOutput, d_LayerError, d_weight, Net->Gain, units, prevUnits);
        AdjustWeightsKernel<<<numBlocks, blockSize>>>(d_prevlayerOutput, d_LayerError, d_weight, d_dweight, Net->Eta, Net->Alpha, units, prevUnits);
        // printf("jjj");
        // fflush(stdout);
        hipMemcpy(Net->Layer[l]->Weight, d_weight, size * sizeof(REAL), hipMemcpyDeviceToHost);
        hipMemcpy(Net->Layer[l]->dWeight, d_dweight, size * sizeof(REAL), hipMemcpyDeviceToHost);

        hipFree(d_prevlayerOutput);
        hipFree(d_prevLayerError);
        hipFree(d_weight);
        hipFree(d_dweight);
        hipFree(d_LayerError);
    }
}
