#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include.h"

#define TILE_SIZE 16

__global__ void PropagateLayerKernel(REAL* layerOutput, REAL* nextLayerOutput, REAL* weight, const REAL gain, const int units, const int nextUnits)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < nextUnits) {
        REAL sum = 0;
        for (int j = 0; j < units; j++) {
            sum += weight[i * units + j] * layerOutput[j];
        }
        nextLayerOutput[i] = 1 / (1 + exp(-gain * sum));
    }
}


void PropagateNetCUDA(NET *Net, NET *Net_d, int NUM_LAYERS)
{
    int blockSize = TILE_SIZE;

    for (int l = 0; l < NUM_LAYERS - 1; l++)
    {
        int units = Net->Layer[l]->Units;
        int nextUnits = Net->Layer[l + 1]->Units;
        int numBlocks = (nextUnits + blockSize - 1) / blockSize;

        int size = nextUnits * units;
    
        REAL *d_weight, *d_layerOutput, *d_nextLayerOutput;
        hipMalloc((REAL**)&d_weight, size * sizeof(REAL));
        hipMalloc((REAL**)&d_layerOutput, units * sizeof(REAL));
        hipMalloc((REAL**)&d_nextLayerOutput, nextUnits * sizeof(REAL));

        hipMemcpy(d_layerOutput, Net->Layer[l]->Output, units * sizeof(REAL), hipMemcpyHostToDevice);
        hipMemcpy(d_weight, Net->Layer[l + 1]->Weight, size * sizeof(REAL), hipMemcpyHostToDevice);

        PropagateLayerKernel<<<numBlocks, blockSize>>>(d_layerOutput, d_nextLayerOutput, d_weight, Net->Gain, units, nextUnits);

        hipMemcpy(Net->Layer[l + 1]->Output, d_nextLayerOutput, nextUnits * sizeof(REAL), hipMemcpyDeviceToHost);
            
        hipFree(d_layerOutput);
        hipFree(d_nextLayerOutput);
        hipFree(d_weight);
    }
}
